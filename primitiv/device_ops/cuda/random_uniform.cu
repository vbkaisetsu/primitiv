#include "hip/hip_runtime.h"
#include <primitiv/config.h>

#include <primitiv/cuda_device.h>
#include <primitiv/internal/cuda_utils.h>
#include <primitiv/device_ops/cuda/common.h>

namespace {

__global__ void rand_affine_dev(
    float shift, float scale, std::uint32_t size, float *py) {
  const std::uint32_t i = IDX;
  const float r = py[i] == 1.f ? 0.f : py[i];
  if (i < size) py[i] = r * scale + shift;
}

}  // namespace

namespace primitiv {
namespace devices {

void CUDA::random_uniform_impl(float lower, float upper, Tensor &y) {
  const std::uint32_t size = y.shape().size();
  const std::uint32_t num_blocks = GRID_SIZE(size, dim1_x_);
  const float scale = upper - lower;
  CUDA_CALL(::hipSetDevice(dev_id_));
  CURAND_CALL(::hiprandGenerateUniform(state_->hiprand.get(), MDATA(y), size));
  ::rand_affine_dev<<<num_blocks, dim1_x_>>>(lower, scale, size, MDATA(y));
}

}  // namespace devices
}  // namespace primitiv
